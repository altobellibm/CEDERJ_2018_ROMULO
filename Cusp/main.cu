#include "hip/hip_runtime.h"
//

#include <iostream>
#include <time.h>

#include <cusp/coo_matrix.h>
#include <cusp/print.h>
#include <cusp/transpose.h>
#include <cusp/convert.h>
#include <cusp/array2d.h>
#include <cusp/multiply.h>
#include <cusp/array1d.h>
#include <cusp/functional.h>


#include <cusp/dia_matrix.h>
#include <cusp/monitor.h>
#include <cusp/krylov/gmres.h>
#include <thrust/reduce.h>
#include <cusp/sort.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <cusp/elementwise.h>
#include <thrust/iterator/zip_iterator.h>

//#include <cusp/eigen/lanczos.h>
#include <cusp/krylov/bicgstab.h>
#include <hipsolver.h>
#include <hipsparse.h>

#include "leitorbasenumerica.h"
#include "TimingGPU.cuh"
#include "TimingGPU.cu"

#define CLOCKS_PER_MS (CLOCKS_PER_SEC / 1000)

// error check macros
#define CUSPARSE_CHECK(x) {hipsparseStatus_t _c=x; if (_c != HIPSPARSE_STATUS_SUCCESS) {printf("cusparse fail: %d, line: %d\n", (int)_c, __LINE__); exit(-1);}}

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const float LIMITE_INFERIOR_DELTA = 1e-8;


// convert a linear index to a row index
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T>
{
  T C; // number of columns
  
  __host__ __device__
  linear_index_to_row_index(T C) : C(C) {}

  __host__ __device__
  T operator()(T i)
  {
    return i / C;
  }
};

template <typename T>
	struct reciprocal_my : public thrust::unary_function<T,T>
	{
	  T value;
	  reciprocal_my(T thr) : value(thr) {};
	  __host__ __device__ 
	  T operator()(const T& v) const {
		   return sqrt(T(value) / v);
	  }
	};

template <typename T>
	struct remove_col : public thrust::unary_function<T,T>
	{
	  T value;
	  remove_col(T thr) : value(thr) {};
	  __host__ __device__ 
	  T operator()(const T& v) const {
		  if (v % value == 1)
			  return 0;
		  else
			  return 1;
	  }
	};	
	
template <typename T>
	struct column_by_vector : public thrust::unary_function<T,T>
	{
	  T* data;
	  T col;
	  column_by_vector(T *_data, T _col) : data(_data), col(_col) {};
	  __host__ __device__ 
	  T operator()(const thrust::tuple<int,float>& v) {
		   return data[thrust::get<0>(v) % (int)col] * thrust::get<1>(v);
	 }
	};

template <typename T>
struct diag_mul : public thrust::unary_function<T,T>
{
	  T* data;
	  T col;
	  diag_mul(T *_data, T _col) : data(_data), col(_col) {};
	  __host__ __device__ 
	  T operator()(const thrust::tuple<float,int>& v) {
		   return thrust::get<0>(v) * data[thrust::get<1>(v) / (int)col];
	 }
	};

	
template <typename T>
struct is_true : thrust::unary_function<T, T>
{
	T col;
	
	is_true(T _c): col(_c) {};
	
    __host__ __device__
    bool operator()(const T &x)
    {
        return (x % col) != 0;
    }
};

void printMatrix(int m, int n, const float*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            float Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f  ", name, row+1, col+1, Areg);
        }
		 printf("\n");
    } 
}  
 
bool svd(int M, cusp::array2d<float,cusp::device_memory>& M_denseHM, cusp::array2d<float,cusp::device_memory>& U, cusp::array1d<float,cusp::device_memory>& S){
	
	  thrust::device_ptr<float> dev_ptr = &(M_denseHM.values[0]);
	  float *M_raw_ptr = thrust::raw_pointer_cast(dev_ptr);
		
    // --- device side SVD workspace and matrices
    int work_size = 0;

    int *devInfo;       hipMalloc(&devInfo, sizeof(int));
    float *d_U;         hipMalloc(&d_U, M * M * sizeof(float));
    float *d_V;         hipMalloc(&d_V, M * M * sizeof(float));
    float *d_S;         hipMalloc(&d_S, M *     sizeof(float));

    hipsolverStatus_t stat;

    // --- CUDA solver initialization
    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);

    hipsolverDnSgesvd_bufferSize(solver_handle, M, M, &work_size);

    float *work;    
	  hipMalloc(&work, work_size * sizeof(float));
	
	  hipsolverDnSgesvd(solver_handle, 'A', 'A', M, M, M_raw_ptr, M, d_S, d_U, M, d_V, M, work, work_size, NULL, devInfo);
    
    int devInfo_h = 0;
    hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	
	  thrust::device_ptr<float > dev_ptr_U( d_U );
	  thrust::copy(thrust::device,dev_ptr_U, dev_ptr_U + (M*M), U.values.begin());
	
	  thrust::device_ptr<float > dev_ptr_S( d_S );
	  thrust::copy(thrust::device,dev_ptr_S, dev_ptr_S+M, S.begin());
	
    hipsolverDnDestroy(solver_handle);
	  
    return 1;
}

int main(int argc, char* argv[]){

    if(argc != 2){
        cout <<"Falta parametro"<<endl;
        return 1;
    }

    std::string strFile = argv[1];
	
	  //std::cout << strFile << std::endl; 
    
	  BASE_NUM base = BASE_NUM();
    LeitorBaseNumerico::obterDadoArquivo(strFile, base);
    	
	  if (base.getErro()) 
	  	return 1;

    cusp::coo_matrix<unsigned long long,float,cusp::device_memory> F;
    
	  //**** matrix F
    std::cout << "Carregando F" << std::endl;
    F = base.getMatrix();
	  std::cout << "Terminado F" << std::endl;
    TimingGPU timer_GPU;
    timer_GPU.StartCounter();
    clock_t Start = clock();
    //std::cout << "F" << std::endl;
  	//cusp::print(F);
   
//	std::cout << "F: n= "<< F.num_rows <<", m= " << F.num_cols <<std::endl;	
	  
    // **** compute the transpose
    cusp::coo_matrix<unsigned long long,float,cusp::device_memory> FT;
    cusp::transpose(F, FT);
	
    //std::cout << "Transposta" << std::endl;													
	  //cusp::print(FT);
	
  	// **** FALTA calcular marginal F da linha e coluna

  	// fr
  	cusp::array1d<int,cusp::device_memory> index_sum_r(F.num_rows);
	  cusp::array1d<float,cusp::device_memory> marginal_sum_r(F.num_rows);
	
  	thrust::reduce_by_key(F.row_indices.begin(), F.row_indices.end(), F.values.begin(), index_sum_r.begin(), marginal_sum_r.begin());

	  //cusp::print(marginal_sum_r);
	
	  // inversa
 	  cusp::array1d<float,cusp::device_memory> fr_inv(F.num_rows, 0);
  	thrust::transform(marginal_sum_r.begin(), marginal_sum_r.end(), fr_inv.begin(), cusp::reciprocal_functor<float>());

  	//cusp::print(fr_inv);

  	// fc
  	cusp::array1d<int,cusp::device_memory> index_sum_c(F.num_cols);
  	cusp::array1d<float,cusp::device_memory> marginal_sum_c(F.num_cols);
	
  	cusp::sort_by_row(F.column_indices, F.row_indices, F.values);

  	//cusp::print(F);
	
  	thrust::reduce_by_key(F.column_indices.begin(), F.column_indices.end(), F.values.begin(), index_sum_c.begin(), marginal_sum_c.begin());
	
	  //cusp::print(marginal_sum_c);
	
  	cusp::array1d<float,cusp::device_memory> fc_inv(F.num_cols, 0);
    thrust::transform(marginal_sum_c.begin(), marginal_sum_c.end(), fc_inv.begin(), cusp::reciprocal_functor<float>());

  	cusp::sort_by_row_and_column(F.row_indices, F.column_indices, F.values);
  	//cusp::print(F);
	
  	//**** Dc
  	// allocate storage for (F.num_cols,F.num_cols) matrix with F.num_cols nonzeros in 1 diagonals
  	cusp::dia_matrix<unsigned long long,float,cusp::device_memory> Dc_inv(F.num_cols,F.num_cols,F.num_cols,1);
    // initialize diagonal offsets
  	Dc_inv.diagonal_offsets[0] = 0;

  	auto diag_c = Dc_inv.values.column(0);
	
  	thrust::copy(thrust::device,fc_inv.begin(), fc_inv.end(), diag_c.begin());
 
  	//cusp::print(Dc_inv);

  	// **** Dr
  	// allocate storage for (F.num_rows,F.num_rows) matrix with F.num_rows nonzeros in 1 diagonals
  	cusp::dia_matrix<unsigned long long,float,cusp::device_memory> Dr_inv(F.num_rows,F.num_rows,F.num_rows,1);
  	// initialize diagonal offsets
  	Dr_inv.diagonal_offsets[0] = 0;
	
  	auto diag_r = Dr_inv.values.column(0);

  	thrust::copy(thrust::device,fr_inv.begin(), fr_inv.end(), diag_r.begin());	
/*
  	cusp::csr_matrix<int,float,cusp::device_memory> DR(F.num_rows,F.num_rows,F.num_rows);
  	cusp::array1d<int,cusp::device_memory> rows(F.num_rows);
  	thrust::sequence(rows.begin(), rows.end(),0);
    thrust::copy(thrust::device,rows.begin(), rows.end(), DR.row_offsets.begin());
  	DR.row_offsets[F.num_rows] = F.num_rows;
  	thrust::copy(thrust::device,rows.begin(), rows.end(), DR.column_indices.begin());
    thrust::copy(thrust::device,fr_inv.begin(), fr_inv.end(), DR.values.begin());

	
  	cusp::csr_matrix<int,float,cusp::device_memory> DC(F.num_cols,F.num_cols,F.num_cols);
  	cusp::array1d<int,cusp::device_memory> cols(F.num_cols);
  	thrust::sequence(cols.begin(), cols.end(),0);
    thrust::copy(thrust::device,cols.begin(), cols.end(), DC.row_offsets.begin());
  	DC.row_offsets[F.num_cols] = F.num_cols;
  	thrust::copy(thrust::device,cols.begin(), cols.end(), DC.column_indices.begin());
    thrust::copy(thrust::device,fc_inv.begin(), fc_inv.end(), DC.values.begin());
*/	
    //cusp::print(DR);
  	//cusp::print(DC);
	
	/*
	cusp::coo_matrix<unsigned long long,float,cusp::host_memory> FT_H(FT);
    cusp::coo_matrix<unsigned long long,float,cusp::host_memory> Dr_inv_H(Dr_inv);
    cusp::coo_matrix<unsigned long long,float,cusp::host_memory> F_H(F);
    cusp::coo_matrix<unsigned long long,float,cusp::host_memory> Dc_inv_H(Dc_inv);
    cusp::coo_matrix<unsigned long long,float,cusp::host_memory> M1_H;
    cusp::coo_matrix<unsigned long long,float,cusp::host_memory> M2_H;
	cusp::coo_matrix<unsigned long long,float,cusp::host_memory> M_H;
    */
    cusp::coo_matrix<unsigned long long,float,cusp::device_memory> M1;
    cusp::coo_matrix<unsigned long long,float,cusp::device_memory> M2;
    cusp::coo_matrix<unsigned long long,float,cusp::device_memory> M;
	  
    
    
    std::cout << "Inicio da Multiplica��o" << std::endl;
  	//cusp::multiply(FT,DR,M1);
    cusp::multiply(FT,Dr_inv,M1);
  	//cusp::multiply(FT_H,Dr_inv_H,M1_H);
    
    //cusp::sort_by_row_and_column(M1.row_indices, M1.column_indices, M1.values);
  	//std::cout << "M1(" << M1.num_rows << " , " << M1.num_cols << " ) = "  << M1.num_entries << std::endl;
	  std::cout << "2a parte da Multiplica��o" << std::endl;
  	//std::cin.ignore();
  	//cusp::print(M1);
  	cusp::multiply(M1,F,M2);
	//cusp::multiply(M1_H,F_H,M2_H);
    
  	//std::cout << "M2(" << M2.num_rows << " , " << M2.num_cols << " ) = "  << M2.num_entries << std::endl;
  	//std::cin.ignore();
    
	std::cout << "3a parte da Multiplica��o" << std::endl;
    cusp::multiply(M2,Dc_inv,M);
    //cusp::multiply(M2_H,Dc_inv_H,M_H);
    
    //cusp::coo_matrix<unsigned long long,float,cusp::device_memory> M(M_H);
    
  	cusp::array2d<float,cusp::device_memory> M_denseHM;
    
    cusp::convert(M,M_denseHM);
	
  	//cusp::print(M_denseHM);

  	cusp::array2d<float,cusp::device_memory> U(F.num_cols,F.num_cols);
  	cusp::array1d<float,cusp::device_memory> S(F.num_cols);
  	
//  std::cout << "M_denseHM: n= "<< M_denseHM.num_rows <<", m= " << M_denseHM.num_cols <<std::endl;
    std::cout << "Inicio da Decomposicao" << std::endl;
  	svd(F.num_cols,M_denseHM, U, S);
/*	
    S[0] = 0.826388;
    S[1] = 0.789907;
    S[2] = 0.210093;
    S[3] = 0.173612;

	//cusp::print(D);

    U.values[0] = 0.478171;
    U.values[1] = 0.0710447;
    U.values[2] = 0.0710447;
    U.values[3] = 0.478171;
    U.values[4] = -0.0985857;
    U.values[5] = -0.551342;
    U.values[6] = -0.551342;
    U.values[7] = -0.0985857;
    U.values[8] = -0.379585;
    U.values[9] = 0.480297;
    U.values[10] = 0.480297;
    U.values[11] = -0.379585;
    U.values[12] = 0.624277;
    U.values[13] = 0.0823854;
    U.values[14] = -0.0823854;
    U.values[15] = -0.624277;
    U.values[16] = -0.440523;
    U.values[17] = 0.433206;
    U.values[18] = -0.433206;
    U.values[19] = 0.440523;
    U.values[20] = -0.183754;
    U.values[21] = -0.515592;
    U.values[22] = 0.515592;
    U.values[23] = 0.183754;

  	std::cout << "X" << std::endl;
    //cusp::print(U);
  	std::cout << "V" << std::endl;
  	//cusp::print(S);
*/	
  	//**************************************
    
    std::cout << "Fim da Decomposicao" << std::endl;
    cusp::array1d<float,cusp::device_memory> rho(S.size()-1);
    thrust::transform(S.begin()+1, S.end(), rho.begin(), cusp::sqrt_functor<float>());
  	//std::cout << "rho" << std::endl;
  	//cusp::print(rho);

  	cusp::array2d<float,cusp::device_memory> X(U.num_rows,U.num_cols-1);

  	cusp::array1d<int,cusp::device_memory> index(F.num_cols*(F.num_cols-1));
  	thrust::sequence(index.begin(), index.end(),0);	
  	thrust::copy_if( U.values.begin(),  U.values.end(), index.begin(),  X.values.begin(), is_true<int>(F.num_cols));
	
    //std::cout << "X" << std::endl;
  	//cusp::print(X);
	
	
	  //**************************************

    cusp::array2d<float,cusp::device_memory> x_sqr;
    cusp::elementwise(X, X, x_sqr,  thrust::multiplies<float>());

    //std::cout << "X_sqr" << std::endl;
    //cusp::print(x_sqr);
	
  	//**************************************
	
    //thrust::copy(thrust::device,marginal_sum_c.begin(), marginal_sum_c.end(), off_c.begin());
	
  	cusp::array2d<float,cusp::device_memory> T(X.num_rows,X.num_cols);
	
  	cusp::array1d<int,cusp::device_memory> index_X(X.num_rows*X.num_cols);
  	thrust::sequence(index_X.begin(), index_X.end(),0);
  	//cusp::print(index_X);
	
  	thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(x_sqr.values.begin(), index_X.begin())), thrust::make_zip_iterator(thrust::make_tuple(x_sqr.values.end(), index_X.end())), T.values.begin(), diag_mul<float>(thrust::raw_pointer_cast(marginal_sum_c.data()),(float)X.num_cols));
	
  	//std::cout << "T" << std::endl;
  	//cusp::print(T);

  	//**************************************
  	cusp::array1d<int,cusp::device_memory> index_sum_t(T.num_cols);
  	cusp::array1d<float,cusp::device_memory> marginal_sum_t(T.num_cols);
	
  	cusp::array2d<float,cusp::device_memory> T_T(T.num_cols,T.num_rows);
    cusp::transpose(T, T_T);
	
  	thrust::reduce_by_key(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(T.num_rows)), thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(T.num_rows)) + (T.num_rows*T.num_cols), T_T.values.begin(), index_sum_t.begin(), marginal_sum_t.begin(), thrust::equal_to<int>(),thrust::plus<float>());
	
  	//std::cout << "cc" << std::endl;
  	//cusp::print(marginal_sum_t);
	
  	cusp::array1d<float,cusp::device_memory> marginal_sum_t_ft(T.num_cols);
	
  	thrust::transform(marginal_sum_t.begin(), marginal_sum_t.end(), marginal_sum_t_ft.begin(), reciprocal_my<float>(float(F.num_entries)));

  	//cusp::print(marginal_sum_t_ft);
	
	
  	//**************************************
  	cusp::array2d<float,cusp::device_memory> x_normed(X.num_rows,X.num_cols);
		
	  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(index_X.begin(), X.values.begin())), thrust::make_zip_iterator(thrust::make_tuple(index_X.end(), X.values.end())), x_normed.values.begin(), column_by_vector<float>(thrust::raw_pointer_cast(marginal_sum_t_ft.data()),(float)X.num_cols));

    //  std::cout << "x_normed" << std::endl;
  	//  cusp::print(x_normed);
	
  	//**************************************
  	cusp::array2d<float,cusp::device_memory> x_project(X.num_rows,X.num_cols);
	
  	thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(index_X.begin(), x_normed.values.begin())), thrust::make_zip_iterator(thrust::make_tuple(index_X.end(), x_normed.values.end())), x_project.values.begin(), column_by_vector<float>(thrust::raw_pointer_cast(rho.data()),(float)X.num_cols));

    //  std::cout << "x_project" << std::endl;
  	//  cusp::print(x_project);
   
//  std::cout << "X_project: n= "<< x_project.num_rows <<", m= " << x_project.num_cols <<std::endl;
    std::cout << "GPU Timing = " << timer_GPU.GetCounter() << " ms" << std::endl;
    std::cout << "Time Difference: " << (float)((clock() - Start )/ CLOCKS_PER_MS) / (float) 1000 << endl;
    return 0;
}
